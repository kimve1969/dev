/************************************************************************
Copyright:      "Surgutneftegas" PJSC
Autors:         Created by Kim V.E. 2023/08/24
Target:         For education
Annotation:     Index of threads & blocks

*************************************************************************/

#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<omp.h>


__global__ void VecAdd(int* pA){
        // index of thread by block
        int ind_tb = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
        // index of block by grid
        int ind_bg = blockIdx.x + gridDim.x*blockIdx.y + gridDim.x*gridDim.y*blockIdx.z;
        // index of tread by grid
        int ind_tg = ind_tb + blockDim.x*blockDim.y*blockDim.z*ind_bg;

        printf("bx.by.bz:tx.ty.tz\t%d.%d.%d:%d.%d.%d\tblock of grid: %d, thread of block: %d, thread of grid: %d\n",blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, ind_bg, ind_tb, ind_tg);
}

/*
   Ким В.Е.
   Если компилятор видит, что память на девайсе не используется (т.е. или не выделяется, или выделяется и удаляется до вызовв кернела) то сам кернел не вызывается!!! В этом есть логика, т.к. зачем что-то исполнять если реузльтат не забирается....

*/
__host__ int main(int argc, char* argv[]){
        if(argc < 7){
                std::cout<<"You should run: "<<argv[0]<<" xg yg zg xb yb zb\n\txg yg zg - x y z of grid\n\txb yb zb - x y z of block\n\texample: "<<argv[0]<<" 16 1 1 1024 1 1\n";
                return -1;
        }

        //std::cout<<"Start"<<std::endl;
        dim3 BpG(std::stoi(argv[1]),std::stoi(argv[2]),std::stoi(argv[3]));
        dim3 TpB(std::stoi(argv[4]),std::stoi(argv[5]),std::stoi(argv[6]));

        int* d_A;
        hipMalloc((void**)&d_A, sizeof(int));

        // Call kernel
        //std::cout<<"\ncomputation on GPU...\n";
        VecAdd<<<BpG, TpB>>>(d_A);

        hipEvent_t syncEvent;
        hipEventCreate(&syncEvent);
        hipEventRecord(syncEvent);
        hipEventSynchronize(syncEvent);

        //std::cout<<"\ndone\n";

        // Free cuda resources
        hipEventDestroy(syncEvent);
        hipFree(d_A);

        //std::cout<<"End"<<std::endl;
        return 0;
}