/************************************************************************
Copyright:      "Surgutneftegas" PJSC
Autors:         Created by Kim V.E. 2023/08/25
Target:         For education
Annotation:     Timing steps calulation & copy

*************************************************************************/

#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<assert.h>
#include<omp.h>


__global__ void VecAdd(int* pA, int* pB, int *pC){
        int id_b = (blockIdx.x + 1) * (blockIdx.y + 1) * (blockIdx.z + 1);
        int id_t = (threadIdx.x + 1) * (threadIdx.y + 1) * (threadIdx.z + 1);
        int indx = (id_b -1) * (blockDim.x * blockDim.y * blockDim.z) + (id_t - 1);
        //printf("%d, ",indx);
        pC[indx] = pA[indx] + pB[indx];
}

/*
   Ким В.Е.
   Аннотация:
*/
__host__ int main(int argc, char* argv[]){
        if(argc < 3){
                std::cout<<"You should run: "<<argv[0]<<" num_blocks num_threads\n\tnum_blocks\tnumber blocks in grid\n\tnum_threads\tnumber threads in block\n";
                return -1;
        }

        std::cout<<"Start"<<std::endl;

        double t0{0}, t1{0}, t2{0}, t3{0};
        const int B{std::stoi(argv[1])}, T{std::stoi(argv[2])};

        dim3 BpG(B,1,1), TpB(T,1,1);
        std::cout<<"grid dimention: "<<BpG.x<<"."<<BpG.y<<"."<<BpG.z<<std::endl;
        std::cout<<"block dimention: "<<TpB.x<<"."<<TpB.y<<"."<<TpB.z<<std::endl;

        long nelements = (BpG.x * BpG.y * BpG.z) * (TpB.x * TpB.y * TpB.z);
        std::cout<<"number elements: "<<nelements<<std::endl;

        int* h_A = new int[nelements];
        int* h_B = new int[nelements];
        int* h_C = new int[nelements];

        for(int i=0; i<nelements; ++i){
                h_A[i] = i*2;
                h_B[i] = i*3;
        }

        // Cuda initialization
        int* d_dummy;
        t0 = omp_get_wtime();
        hipMalloc((void**)&d_dummy, sizeof(int));

        // Allocate A,B,C vectors in device memory
        size_t size = nelements * sizeof(int);

        t1 = omp_get_wtime();
		
        int* d_A;
        hipMalloc((void**)&d_A, size);
        int* d_B;
        hipMalloc(&d_B, size);
        int* d_C;
        hipMalloc(&d_C, size);

        // Copy A,B vectors from host to device
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        t2 = omp_get_wtime();

        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Call kernel
        std::cout<<"\ncomputation on GPU...\n";
        VecAdd<<< BpG /* blocks per grid */, TpB /* threads per block */>>>(d_A, d_B, d_C);

        hipEventRecord(stop);
        // synchronize host & device
        hipEventSynchronize(stop);

        std::cout<<"\ndone\n";

        t3 = omp_get_wtime();
        // Copy C vector from device to host
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        // Free device resources
        float ms_cuda{0};
        hipEventElapsedTime(&ms_cuda, start, stop);
		
		hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // check last element ...
        assert((h_A[nelements-1] + h_B[nelements-1]) == h_C[nelements-1]);
        std::cout<<"Check last element A,B,C vectors: A[ "<<(nelements-1)<<" ] + B[ "<<(nelements-1)<<" ] = C[ "<<(nelements-1)<<" ] : "<<h_A[nelements-1]<<" + "<<h_B[nelements-1]<<" = "<<h_C[nelements-1]<<"\n";

        // Free host memory
        delete [] h_A;
        delete [] h_B;
        delete [] h_C;


        std::cout<<"Time initialization CUDA: "<<(t1-t0)<<" sec"<<std::endl;
        std::cout<<"Timing are common: "<<(t3-t1)<<" sec, calculate: "<<(t3-t2)<<" sec ( "<<(t3-t2)*100.0/(t3-t1)<<" % ), copy: "<<(t2-t1)<<" sec ("<<(t2-t1)*100.0/(t3-t1)<<"%)"<<std::endl;
        std::cout<<"\nCuda time calculate: "<<ms_cuda/1000.0<<" sec\n";

        std::cout<<"End"<<std::endl;
        return 0;
}