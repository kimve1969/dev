/************************************************************************
Copyright:      "Surgutneftegas" PJSC
Autors:         Created by Kim V.E. 2023/08/23
Target:         For education
Annotation:     Initialization device, copy data from host to device, run kernel

*************************************************************************/

#include <hip/hip_runtime.h>
#include<iostream>
#include<omp.h>

__global__ void VecAdd(int* pA, int* pB, int *pC){
	int i = threadIdx.x;
	printf("%d, ",i); // print() add time computation!
	pC[i] = pA[i] + pB[i];
}

/*
   	Ким В.Е.
	Аннотация:
	1. Первый вызов CUDA вызывает инициализацию.
	2. При задании максимального кол-ва потоков в блоке больше чем 1024 (максимально возможное), компилятор не выдает ошибки. Результат неопределенный.

	Дополнительные выводы:
	1. Код по семантике похож на Open CL, но намного проще (не требуется инициализация платформ и и девайсов)
 */
__host__ int main(int arc, char* argv[]){
	std::cout<<"Start"<<std::endl;
	double t0{0}, t1{0}, t2{0}, t3{0};
	const int N = 100; // max threads is 1024;

	int* h_A = new int[N];
	int* h_B = new int[N];
	int* h_C = new int[N];

	for(int i=0; i<N; ++i){
		h_A[i] = i*2;
		h_B[i] = i*3;
	}

	auto prn = [&](int *p, char ch='M'){
		for(int i=0; i<N; ++i){
			std::cout<<ch<<"["<<i<<"] = "<<p[i]<<", ";
		}
		std::cout<<std::endl;
	};

	std::cout<<"\nA:\n";
	prn(h_A, 'A');

	std::cout<<"\nB:\n";
	prn(h_B, 'B');

	// Cuda initialization
	int* d_dummy;
	t0 = omp_get_wtime();
	hipMalloc((void**)&d_dummy, sizeof(int));

	// Allocate A,B,C vectors in device memory
	size_t size = N * sizeof(int);

	t1 = omp_get_wtime();

	int* d_A;
	hipMalloc((void**)&d_A, size);
	int* d_B;
	hipMalloc(&d_B, size);
	int* d_C;
	hipMalloc(&d_C, size);

	// Copy A,B vectors from host to device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	t2 = omp_get_wtime();
	// Call kernel
	std::cout<<"\ncomputation on GPU...\n";
	VecAdd<<<1, N>>>(d_A, d_B, d_C);

	hipEvent_t syncEvent;
	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);
	std::cout<<"\ndone\n";

	t3 = omp_get_wtime();
	// Copy C vector from device to host
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	std::cout<<"\nC:\n";
	prn(h_C, 'C');

	// Free cuda resources
	hipEventDestroy(syncEvent);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free host memory
	delete [] h_A;
	delete [] h_B;
	delete [] h_C;

	std::cout<<"\nTime initialization CUDA: "<<(t1-t0)<<" sec"<<std::endl;
	std::cout<<"Timing are common: "<<(t3-t1)<<" sec, calculate: "<<(t3-t2)<<" sec ( "<<(t3-t2)*100.0/(t3-t1)<<" % ), copy: "<<(t2-t1)<<" sec ("<<(t2-t1)*100.0/(t3-t1)<<"%)"<<std::endl;
	std::cout<<"NOTE: output (using print function) in kernel add many time in computation!!!"<<std::endl;
	std::cout<<"End"<<std::endl;
	return 0;
}
