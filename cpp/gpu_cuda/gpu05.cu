/************************************************************************
Copyright:      "Surgutneftegas" PJSC
Autors:         Created by Kim V.E. 2023/08/30
Target:         For education
Annotation:     Matrix |C|=|A|+|B| and |C|=|A|*|B|

*************************************************************************/

#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<assert.h>
#include<stdexcept>
#include<omp.h>

enum oper_t
{
        ADD,
        MUL
} _oper_t;

enum prn_t
{
        PRINT,
        NOPRINT
} _prn_t;

__global__ void Create2D(void *p2D, int N, double* p1D)
{
        // index of thread by block
        int ind_tb = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
        // index of block by grid
        int ind_bg = blockIdx.x + gridDim.x*blockIdx.y + gridDim.x*gridDim.y*blockIdx.z;
        // index of tread by grid
        int ind_tg = ind_tb + blockDim.x*blockDim.y*blockDim.z*ind_bg;

        if(ind_tg > N-1)
        {
                return;
        }
		
		 //
        double **p = (double**)p2D;

        p[ind_tg] = &p1D[ind_tg*N];
}

__global__ void A_oper_B(void* p2D_A, void* p2D_B, void* p2D_C, int N, oper_t op = ADD){

        double **A = (double**)p2D_A;
        double **B = (double**)p2D_B;
        double **C = (double**)p2D_C;

        int ind_tb = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
        int ind_bg = blockIdx.x + gridDim.x*blockIdx.y + gridDim.x*gridDim.y*blockIdx.z;
        int ind_tg = ind_tb + blockDim.x*blockDim.y*blockDim.z*ind_bg;

        // recalculate global threads index to i-row an d j-col threads
        int i = ind_tg / N;
        int j = ind_tg % N;

        // check bounders of i & j
        if(i > (N-1) || j > (N-1))
        {
                return;
        }

        // |C| = |A|+|B|
        if(op == ADD){
                C[i][j] = A[i][j] + B[i][j];
        }

        // |C| = |A|*|B|
        if(op == MUL)
        {
                //#pragma unroll
                for(int k=0; k<N; ++k)
                {
                        C[i][j] += A[i][k] * B[k][j];
				}
        }
}

__host__ bool is_int(std::string arg)
{
        try
        {
                const int test{ std::stoi(arg) };
        }
        catch(std::invalid_argument const& ex)
        {
                return false;
        }
        catch(std::out_of_range const& ex)
        {
                return false;
        }

        return true;
}

__host__ int main(int argc, char* argv[]){
        std::cout<<"\nStart\n\n";

        // check arguments
        if( (argc < 3 /* mandatory number arguments  */) ||
            (argc >= 2 && ! is_int( argv[1] ) ) ||
            (argc >= 3 && std::string(argv[2]) != "add" && std::string(argv[2]) != "mul") ||
            (argc >= 4 && std::string(argv[3]) != "print" && std::string(argv[3]) != "noprint" ) ||
            (argc >= 5 && ! is_int( argv[4]) )
          )
        {
                // help
                std::cout<<"Matrix operation: |C|=|A|+|B| or |C|=|A|*|B|\n";
                std::cout<<"Run: "<<argv[0]<<" arg1 arg2 [arg3] [arg4]\n\
Where:\n\
\targ1 - dimention of matrix A,B,C, is mandatoty\n\
\targ2 - [ add | mul ] operation by martix, is mandatory\n\
\targ3 - [ print | noprint ], is optional, default print\n\
\targ4 - number of device (GPU) from 0....(N-1) where N - last number GPU, is optional, default 0 (if use docker, the number begin from 0)\n\
Example:\n"
<<argv[0]<<" 4 add\n"
<<argv[0]<<" 1000 mul noprint\n"
<<argv[0]<<" 3000 mul noprint 2\n";

                std::cout<<"\nEnd\n";

                return -1;
        }

        const int    arg_N{std::stoi(argv[1])};
        const oper_t arg_operation{ std::string( argv[2] ) == "add" ? ADD : MUL };
        const prn_t  arg_print{ argc >= 4 && std::string( argv[3] ) == "print" ? PRINT : NOPRINT /* default print */};
        const int    arg_device{ argc >= 5 ? std::stoi( argv[4] )  : 0 /* default device is 0 */};

        hipSetDevice( arg_device );

        double t1{0}, t2{0};

        t1 = omp_get_wtime();

        long nelements = arg_N * arg_N;
        std::cout<<"number elements: "<<nelements<<std::endl;

        // 2-х мерный массив в виде массива указателей на одномерный массив, т.к.
        // потом в пакетах BLAS или cuBLAS потребуется передавать именно НЕПРЕРЫВНЫЕ массивы
        // Есть два способа:
        //      1-ый - высчитывать псеводо-индексы 2-х мерного массива
        //      2-ой - создать доплнительный массив указателей
        //      скорее всего 2-ой будет работать быстрее, т.к. исключаются вычисления псевдо-индексов
        double* h_A = new double[nelements];
        double* h_B = new double[nelements];
        double* h_C = new double[nelements];

        for(int i=0; i<nelements; ++i){
                h_A[i] = i*2.1;
                h_B[i] = i*3.1;
                h_C[i] = 0.0;
        }

        double** h_matrix_A = new double*[arg_N];
        double** h_matrix_B = new double*[arg_N];
        double** h_matrix_C = new double*[arg_N];

        for(int i=0; i<arg_N; ++i){
                h_matrix_A[i] = &h_A[i*arg_N];
                h_matrix_B[i] = &h_B[i*arg_N];
                h_matrix_C[i] = &h_C[i*arg_N];
        }

        auto prn = [ &arg_print ](std::string sinfo, double** M, int N){
                if(arg_print != PRINT)
                {
                        return;
                }

                std::cout<<sinfo;

                for(int i=0; i<N; ++i){
                        for(int j=0; j<N; ++j){
                                std::cout<<M[i][j]<<"\t";
                        }
                        std::cout<<"\n";
                }
        };

        prn("\n\n|A|:\n\n", h_matrix_A, arg_N);
        prn("\n\n|B|:\n\n", h_matrix_B, arg_N);

        // CUDA timing
        float ms[10]{0};
		hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        // Cuda initialization
        int* d_dummy;
        hipMalloc((void**)&d_dummy, sizeof(double));

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms[0], start, stop);

        // Allocate A,B,C vectors in device memory
        size_t size = nelements * sizeof(double);

        // Create arrays of int
        double* d_A;
        hipMalloc((void**)&d_A, size);
        double* d_B;
        hipMalloc(&d_B, size);
        double* d_C;
        hipMalloc(&d_C, size);

        // Create arrays of pointer on int
        void* d_matrix_A;
        hipMalloc( (void**)&d_matrix_A, arg_N*sizeof(double*));
        void* d_matrix_B;
        hipMalloc( (void**)&d_matrix_B, arg_N*sizeof(double*));
        void* d_matrix_C;
        hipMalloc( (void**)&d_matrix_C, arg_N*sizeof(double*));

        // Link arrays of pointer with array of double
        Create2D<<<dim3((arg_N/1024)+1,1,1), dim3(1024,1,1) >>>(d_matrix_A, arg_N, d_A);
        Create2D<<<dim3((arg_N/1024)+1,1,1), dim3(1024,1,1) >>>(d_matrix_B, arg_N, d_B);
        Create2D<<<dim3((arg_N/1024)+1,1,1), dim3(1024,1,1) >>>(d_matrix_C, arg_N, d_C);

        // Copy A,B vectors from host to device
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms[1], start, stop);

        // Call kernel
        std::cout<<"\ncomputation on GPU...\n";
        A_oper_B<<< dim3( arg_N*arg_N/1024+1, 1, 1) /* blocks per grid */, dim3(1024,1,1) /* threads per block */>>>(d_matrix_A, d_matrix_B, d_matrix_C, arg_N, arg_operation);

        // synchronize host & device
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms[2], start, stop);

        std::cout<<"\ndone\n";

        // Copy C vector from device to host
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        prn("\n\n|C|:\n\n", h_matrix_C, arg_N);

        // Free device resources
        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        hipFree(d_matrix_A);
        hipFree(d_matrix_B);
        hipFree(d_matrix_C);

        // check last element ...
        auto prn_check = [&](int i, int j)
		{
                std::cout<<"C[ "<<i<<" ][ "<<j<<" ] = "<<h_matrix_C[i][j]<<"\n";
        };

        std::cout<<"\nCheck results:\n";
        prn_check(0, 1);
        prn_check(arg_N-1, arg_N-1);

        // Free host memory
        delete [] h_A;
        delete [] h_B;
        delete [] h_C;

        t2 = omp_get_wtime();

        std::cout<<"\nOMP common time: "<<(t2-t1)<<" sec.\n";

        std::cout<<"CUDA common time: "<<ms[2]<<" msec. (100%),  init: "<<ms[0]<<" msec. ( "<<ms[0]*100.0/ms[2]<<" % ), copy: "<<ms[1]-ms[0]<<" msec. ( "<<(ms[1]-ms[0])*100.0/ms[2]<<" % ), calc: "<<ms[2]-ms[1]<<" msec. ( "<<(ms[2]-ms[1])*100.0/ms[2] <<" % )\n";

        std::cout<<"End\n"<<std::endl;

        return 0;
}
